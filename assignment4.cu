#include "hip/hip_runtime.h"
/* Vector addition deom on GPU

   To compile: nvcc -o testprog1 testprog1.cu

 */
#include <ctime>
#include <iostream>
#include <string>
#include <cmath>

#include <hip/hip_runtime.h>
#include "lenses.h"
#include "arrayff.hxx"


// Boundaries in physical units on the lens plane
const float WL  = 2.0;
const float XL1 = -WL;
const float XL2 =  WL;
const float YL1 = -WL;
const float YL2 =  WL;

// Source star parameters. You can adjust these if you like - it is
// interesting to look at the different lens images that result
const float rsrc = 0.1;      // radius
const float ldc  = 0.5;      // limb darkening coefficient
const float xsrc = 0.0;      // x and y centre on the map
const float ysrc = 0.0;

// Used to time code. OK for single threaded programs but not for
// multithreaded programs. See other demos for hints at timing CUDA
// code.
double diffclock(clock_t clock1,clock_t clock2)
{
  double diffticks = clock1 - clock2;
  double diffms = (diffticks * 1000) / CLOCKS_PER_SEC;
  return diffms; // Time difference in milliseconds
}

void err_check(hipError_t err){
	if(err != hipSuccess){
		std::cout << "Cuda Error: " << hipGetErrorString(hipGetLastError())<< std::endl;
	}
}

// Kernel that executes on the CUDA device. This is executed by ONE
// stream processor
__global__ void ray_shoot(int *maxX, int *maxY, float *lens_scale, float *xlens, float *ylens, float*eps, int *num_lenses, float *dev_arr)
{
	int threadBlockPos = (blockIdx.x * blockDim.x) + threadIdx.x;

	int y = threadBlockPos / (*maxY);
	int x = threadBlockPos - ((*maxX) * y);

	const float rsrc2 = rsrc * rsrc; 

	float xl = XL1 + x * (*lens_scale);
	float yl = YL1 + y * (*lens_scale); 
	float xs = XL2 + x * (*lens_scale); 
	float ys = YL2 + y * (*lens_scale);

	float dx, dy, dr;
	xs = xl;
	ys = yl;
	for(int p = 0; p < (*num_lenses); ++p){
		dx = xl - xlens[p];
	    dy = yl - ylens[p];
	    dr = dx * dx + dy * dy;
	    xs -= eps[p] * dx / dr;
	    ys -= eps[p] * dy / dr;
	}

	float xd = xs - xsrc; 
	float yd = ys - ysrc; 
	float sep2 = (xd * xd) + (yd * yd); 
	
	if(sep2 < rsrc2){
		float mu = sqrtf(1.0f-sep2/rsrc2); 
		dev_arr[threadBlockPos] = 1.0 - ldc * (1-mu);
	}
}

// main routine that executes on the host
int main(void)
{

	// Set up lensing system configuration - call example_1, _2, _3 or
	// _n as you wish. The positions and mass fractions of the lenses
	// are stored in these arrays
	float* xlens;
	float* ylens;
	float* eps;
	const int nlenses = set_example_2(&xlens, &ylens, &eps);
	std::cout << "# Simulating " << nlenses << " lens system" << std::endl;


	// Pixel size in physical units of the lens image. You can try finer
	// lens scale which will result in larger images (and take more
	// time).
	const float lens_scale = 0.005;

	// Size of the lens image
	const int npixx = static_cast<int>(floor((XL2 - XL1) / lens_scale)) + 1;
	const int npixy = static_cast<int>(floor((YL2 - YL1) / lens_scale)) + 1;
	std::cout << "# Building " << npixx << "X" << npixy << " lens image" << std::endl;

	// Put the lens image in this array
  	Array<float, 2> lensim(npixy, npixx);

	clock_t tstart = clock();

	int total_pixels = npixx * npixy;
	std::cout << "total pixels: " << total_pixels << std::endl;
	int threadsPerBlock = 1024;
	std::cout << "total Threads per block: " << threadsPerBlock << std::endl;
	int numBlocks = (total_pixels + threadsPerBlock - 1) / threadsPerBlock;
	std::cout << "total blocks per grid: " << numBlocks << std::endl;

	//setup the array that will be sent to device for all of the pixels and will eventually be retrieved
	float *arr_lensim = (float*)malloc(sizeof(float)*npixx*npixy);

	for(int y = 0; y < npixy; y++){
		for(int x = 0; x < npixx; x++){
			arr_lensim[(y * npixx) + x] = 0;
		}
	}

	std::cout << "host----: " << eps[0] << std::endl;

	//--------------------------
	//cuda part
	//==========================

	int *dev_npixx;
	int *dev_npixy;
	float *dev_lens_scale;
	float *dev_xlens;
	float *dev_ylens; 
	float *dev_eps;
	int *dev_nlenses;
	float *dev_arr_lensim;

	int size = sizeof(float) * nlenses;

	hipMalloc((void**)&dev_npixx, sizeof(int));
	hipMalloc((void**)&dev_npixy, sizeof(int));
	hipMalloc((void**)&dev_lens_scale, sizeof(float));
	hipMalloc((void**)&dev_xlens, size);
	hipMalloc((void**)&dev_ylens, size);
	hipMalloc((void**)&dev_eps, size);
	hipMalloc((void**)&dev_nlenses, sizeof(int));
	hipMalloc((void**)&dev_arr_lensim, sizeof(float)*npixx*npixy);

	hipMemcpy(dev_npixx, &npixx, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_npixy, &npixy, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_lens_scale, &lens_scale, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_xlens, xlens, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_ylens, ylens, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_eps, eps, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_nlenses, &nlenses, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr_lensim, arr_lensim, sizeof(float)*npixx*npixy, hipMemcpyHostToDevice);

	//=====================================
	//=====================================
	//Need to create a new array on both host and device memory and then copy from device the new values and then convert it back to the Array object type so that we can use dump_array();
	
	ray_shoot<<<numBlocks, threadsPerBlock>>>(dev_npixx, dev_npixy, dev_lens_scale, dev_xlens, dev_ylens, dev_eps, dev_nlenses, dev_arr_lensim);

	hipMemcpy(arr_lensim, dev_arr_lensim, sizeof(float)*npixx*npixy, hipMemcpyDeviceToHost);


	for(int y = 0; y < npixy; y++){
		for(int x = 0; x < npixx; x++){
			if(arr_lensim[(y*npixy)+x] != 0.0){
				lensim(y, x) = arr_lensim[(y*npixx) + x];
			}
		}
	}

	hipFree(dev_xlens); 
	hipFree(dev_ylens); 
	hipFree(dev_eps);
	hipFree(dev_arr_lensim);

	clock_t tend = clock();
	double tms = diffclock(tend, tstart);
	std::cout << "# Time elapsed: " << tms << " ms " << std::endl;

	// Write the lens image to a FITS formatted file. You can view this
	// image file using ds9
	dump_array<float, 2>(lensim, "lens2.fit");

	delete[] xlens;
	delete[] ylens;
	delete[] eps;
}
